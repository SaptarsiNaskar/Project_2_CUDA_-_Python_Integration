#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void haversine_distance_kernel(int size, const double *x1,const double *y1,
    const double *x2,const double *y2, double *dist)
{
 //use any references to compute haversine distance bewtween (x1,y1) and (x2,y2), given in vectors/arrays
 //e.g., https://stackoverflow.com/questions/27928/calculate-distance-between-two-latitude-longitude-points-haversine-formula
}


void run_kernel(int size, const double *x1,const double *y1, const double *x2,const double *y2, double *dist)
   
{
  dim3 dimBlock(1024);
  printf("in run_kernel dimBlock.x=%d\n",dimBlock.x);

  dim3 dimGrid(ceil((double)size / dimBlock.x));
  
  haversine_distance_kernel<<<dimGrid, dimBlock>>>
    (size,x1,y1,x2,y2,dist);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::stringstream strstr;
    strstr << "run_kernel launch failed" << std::endl;
    strstr << "dimBlock: " << dimBlock.x << ", " << dimBlock.y << std::endl;
    strstr << "dimGrid: " << dimGrid.x << ", " << dimGrid.y << std::endl;
    strstr << hipGetErrorString(error);
    throw strstr.str();
  }
}
